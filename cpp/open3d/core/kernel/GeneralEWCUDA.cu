#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include "open3d/core/Dispatch.h"
#include "open3d/core/Dtype.h"
#include "open3d/core/MemoryManager.h"
#include "open3d/core/SizeVector.h"
#include "open3d/core/Tensor.h"
#include "open3d/core/hashmap/Hashmap.h"
#include "open3d/core/kernel/CUDALauncher.cuh"
#include "open3d/core/kernel/GeneralEW.h"
#include "open3d/core/kernel/GeneralEWMacros.h"
#include "open3d/core/kernel/GeneralEWSharedImpl.h"
#include "open3d/core/kernel/GeneralIndexer.h"
#include "open3d/utility/Console.h"

namespace open3d {
namespace core {
namespace kernel {

struct Coord3i {
    OPEN3D_HOST_DEVICE Coord3i(int x, int y, int z) : x_(x), y_(y), z_(z) {}
    bool OPEN3D_HOST_DEVICE operator==(const Coord3i& other) const {
        return x_ == other.x_ && y_ == other.y_ && z_ == other.z_;
    }

    int64_t x_;
    int64_t y_;
    int64_t z_;
};

void CUDATSDFTouchKernel(const std::unordered_map<std::string, Tensor>& srcs,
                         std::unordered_map<std::string, Tensor>& dsts) {
    static std::vector<std::string> src_attrs = {
            "points",
            "voxel_size",
            "resolution",
    };

    for (auto& k : src_attrs) {
        if (srcs.count(k) == 0) {
            utility::LogError(
                    "[CUDATSDFTouchKernel] expected Tensor {} in srcs, but "
                    "did not receive",
                    k);
        }
    }

    Tensor pcd = srcs.at("points");
    float voxel_size = srcs.at("voxel_size").Item<float>();
    int64_t resolution = srcs.at("resolution").Item<int64_t>();
    float block_size = voxel_size * resolution;

    float sdf_trunc = srcs.at("sdf_trunc").Item<float>();

    Device device = pcd.GetDevice();

    int64_t n = pcd.GetLength();
    float* pcd_ptr = static_cast<float*>(pcd.GetDataPtr());

    Tensor block_coordi({8 * n, 3}, Dtype::Int32, device);
    int* block_coordi_ptr = static_cast<int*>(block_coordi.GetDataPtr());
    Tensor count(std::vector<int>{0}, {}, Dtype::Int32, device);
    int* count_ptr = static_cast<int*>(count.GetDataPtr());

    CUDALauncher::LaunchGeneralKernel(n, [=] OPEN3D_DEVICE(
                                                 int64_t workload_idx) {
        float x = pcd_ptr[3 * workload_idx + 0];
        float y = pcd_ptr[3 * workload_idx + 1];
        float z = pcd_ptr[3 * workload_idx + 2];

        int xb_lo = static_cast<int>(floor((x - sdf_trunc) / block_size));
        int xb_hi = static_cast<int>(floor((x + sdf_trunc) / block_size));
        int yb_lo = static_cast<int>(floor((y - sdf_trunc) / block_size));
        int yb_hi = static_cast<int>(floor((y + sdf_trunc) / block_size));
        int zb_lo = static_cast<int>(floor((z - sdf_trunc) / block_size));
        int zb_hi = static_cast<int>(floor((z + sdf_trunc) / block_size));

        for (int xb = xb_lo; xb <= xb_hi; ++xb) {
            for (int yb = yb_lo; yb <= yb_hi; ++yb) {
                for (int zb = zb_lo; zb <= zb_hi; ++zb) {
                    int idx = atomicAdd(count_ptr, 1);
                    block_coordi_ptr[3 * idx + 0] = xb;
                    block_coordi_ptr[3 * idx + 1] = yb;
                    block_coordi_ptr[3 * idx + 2] = zb;
                }
            }
        }
    });

    int total_block_count = count.Item<int>();
    block_coordi = block_coordi.Slice(0, 0, total_block_count);
    core::Hashmap pcd_block_hashmap(total_block_count, core::Dtype::Int32,
                                    core::Dtype::Int32, {3}, {1}, device);
    core::Tensor block_addrs, block_masks;
    pcd_block_hashmap.Activate(block_coordi.Slice(0, 0, count.Item<int>()),
                               block_addrs, block_masks);
    dsts.emplace("block_coords", block_coordi.IndexGet({block_masks}));
}

void GeneralEWCUDA(const std::unordered_map<std::string, Tensor>& srcs,
                   std::unordered_map<std::string, Tensor>& dsts,
                   GeneralEWOpCode op_code) {
    switch (op_code) {
        case GeneralEWOpCode::Unproject:
            CUDAUnprojectKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFTouch:
            CUDATSDFTouchKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFIntegrate:
            CUDATSDFIntegrateKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFPointExtraction:
            CUDAPointExtractionKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::TSDFMeshExtraction:
            CUDAMeshExtractionKernel(srcs, dsts);
            break;
        case GeneralEWOpCode::RayCasting:
            utility::LogError("[RayCasting] Unimplemented.");
            break;
        default:
            break;
    }
}

}  // namespace kernel
}  // namespace core
}  // namespace open3d
